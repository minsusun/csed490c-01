#include "hip/hip_runtime.h"
// Given a list (lst) of length n
// Output its prefix sum = {lst[0], lst[0] + lst[1], lst[0] + lst[1] + ...
// +
// lst[n-1]}

#include <gputk.h>

#define BLOCK_SIZE 1024 //@@ You can change this

#define gpuTKCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      gpuTKLog(ERROR, "Failed to run stmt ", #stmt);                         \
      gpuTKLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

__global__ void add_block_sum(float *input, float *output, int len){
  int bid = blockIdx.x;
  int idx = bid * blockDim.x + threadIdx.x;

  if (bid > 0 && idx < len)
    output[idx] += input[bid - 1];
}

__global__ void scan(float *input, float *output, float *aux, int len) {
  //@@ Modify the body of this function to complete the functionality of
  //@@ the scan on the device
  //@@ You may need multiple kernel calls; write your kernels before this
  //@@ function and call them from here
  __shared__ float T[BLOCK_SIZE];

  int tid = threadIdx.x;
  int base_idx = blockIdx.x * blockDim.x;

  // import data
  if (base_idx + tid < len)
    T[tid] = input[base_idx + tid];
  else
    T[tid] = 0.0f;

  // pre-scan step
  for (unsigned stride = 1; stride <= blockDim.x; stride *= 2) {
    __syncthreads();

    int index = (tid + 1) * 2 * stride - 1;

    if (index < BLOCK_SIZE)
      T[index] += T[index - stride];
  }

  // post-scan step
  for (int stride = BLOCK_SIZE / 4; stride > 0; stride /= 2) {
    __syncthreads();

    int index = (tid + 1) * stride * 2 - 1;

    if (index + stride < BLOCK_SIZE)
      T[index + stride] += T[index];
  }

  // export data
  __syncthreads();
  if (base_idx + tid < len)
    output[base_idx + tid] = T[tid];
  if (aux != NULL && tid == blockDim.x - 1)
    aux[blockIdx.x] = T[tid];
}

int main(int argc, char **argv) {
  gpuTKArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output list
  float *deviceInput;
  float *deviceOutput;
  int numElements; // number of elements in the list

  float *aux_1_in;
  float *aux_1_out;
  float *aux_2_in;
  float *aux_2_out;
  int numAux1;
  int numAux2;

  args = gpuTKArg_read(argc, argv);

  gpuTKTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (float *)gpuTKImport(gpuTKArg_getInputFile(args, 0), &numElements);
  hostOutput = (float *)malloc(numElements * sizeof(float));
  gpuTKTime_stop(Generic, "Importing data and creating memory on host");

  gpuTKLog(TRACE, "The number of input elements in the input is ",
        numElements);

  gpuTKTime_start(GPU, "Allocating GPU memory.");
  gpuTKCheck(hipMalloc((void **)&deviceInput, numElements * sizeof(float)));
  gpuTKCheck(hipMalloc((void **)&deviceOutput, numElements * sizeof(float)));

  numAux1 = (numElements - 1) / BLOCK_SIZE + 1;
  numAux2 = (numAux1 - 1) / BLOCK_SIZE + 1;
  gpuTKCheck(hipMalloc((void **)&aux_1_in, numAux1 * sizeof(float)));
  gpuTKCheck(hipMalloc((void **)&aux_1_out, numAux1 * sizeof(float)));
  gpuTKCheck(hipMalloc((void **)&aux_2_in, numAux2 * sizeof(float)));
  gpuTKCheck(hipMalloc((void **)&aux_2_out, numAux2 * sizeof(float)));
  gpuTKTime_stop(GPU, "Allocating GPU memory.");

  gpuTKTime_start(GPU, "Clearing output memory.");
  gpuTKCheck(hipMemset(deviceOutput, 0, numElements * sizeof(float)));
  gpuTKTime_stop(GPU, "Clearing output memory.");

  gpuTKTime_start(GPU, "Copying input memory to the GPU.");
  gpuTKCheck(hipMemcpy(deviceInput, hostInput, numElements * sizeof(float),
                     hipMemcpyHostToDevice));
  gpuTKTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 dimBlock(BLOCK_SIZE, 1, 1);

  dim3 dimGrid_1(numAux1, 1, 1);
  dim3 dimGrid_2(numAux2, 1, 1);
  dim3 dimGrid_3((numAux2 - 1) / BLOCK_SIZE, 1, 1);
  dim3 dimGrid_4(numAux2, 1, 1);
  dim3 dimGrid_5(numAux1, 1, 1);

  gpuTKTime_start(Compute, "Performing CUDA computation");
  
  //@@ Modify this to complete the functionality of the scan
  //@@ on the deivce
  scan<<<dimGrid_1, dimBlock, BLOCK_SIZE * sizeof(float)>>>(deviceInput, deviceOutput, aux_1_in, numElements);
  hipDeviceSynchronize();
  scan<<<dimGrid_2, dimBlock, BLOCK_SIZE * sizeof(float)>>>(aux_1_in, aux_1_out, aux_2_in, numAux1);
  hipDeviceSynchronize();
  scan<<<dimGrid_3, dimBlock, BLOCK_SIZE * sizeof(float)>>>(aux_2_in, aux_2_out, NULL, numAux2);
  hipDeviceSynchronize();
  add_block_sum<<<dimGrid_4, dimBlock>>>(aux_2_out, aux_1_out, numAux1);
  hipDeviceSynchronize();
  add_block_sum<<<dimGrid_5, dimBlock>>>(aux_1_out, deviceOutput, numElements);
  hipDeviceSynchronize();

  gpuTKTime_stop(Compute, "Performing CUDA computation");

  gpuTKTime_start(Copy, "Copying output memory to the CPU");
  gpuTKCheck(hipMemcpy(hostOutput, deviceOutput, numElements * sizeof(float),
                     hipMemcpyDeviceToHost));
  gpuTKTime_stop(Copy, "Copying output memory to the CPU");

  gpuTKTime_start(GPU, "Freeing GPU Memory");
  hipFree(deviceInput);
  hipFree(deviceOutput);

  hipFree(aux_1_in);
  hipFree(aux_1_out);
  hipFree(aux_2_in);
  hipFree(aux_2_out);

  gpuTKTime_stop(GPU, "Freeing GPU Memory");

  gpuTKSolution(args, hostOutput, numElements);

  free(hostInput);
  free(hostOutput);

  return 0;
}
