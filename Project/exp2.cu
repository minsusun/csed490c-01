
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <cassert>
#include <fstream>
#include <algorithm>
#include <ctime>

#define TILE_WIDTH 32
#define THREADS_PER_BLOCK 512
using namespace std;

#define CUDACheck(stmt)                                                     \
  do {                                                                      \
    hipError_t err = stmt;                                                 \
    if (err != hipSuccess) {                                               \
      cout << "Failed to run " << #stmt << endl;                            \
      cout << "Got CUDA error ...  " << hipGetErrorString(err) << endl;    \
      return -1;                                                            \
    }                                                                       \
  } while (0)

string name = "CUDA(pinned memory/matrix distance calculation/odd-even sort)";

double interval(clock_t *p) {
  clock_t t = clock();
  double result = double(t - *p) / CLOCKS_PER_SEC * 1000;
  *p = t;
  return result;
}

__global__ void matrixDistance(double *X, double *X_t, double *D, int *idxMat, int N, int dim) {
  __shared__ double ds_A[TILE_WIDTH][TILE_WIDTH];
  __shared__ double ds_B[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int Row = by * blockDim.y + ty;
  int Col = bx * blockDim.x + tx;
  
  double Cvalue = 0.0;

  for (int phase = 0; phase < (dim - 1) / TILE_WIDTH + 1; phase++) {
    if (Row < N && phase * TILE_WIDTH + tx < dim) ds_A[ty][tx] = X[Row * dim + phase * TILE_WIDTH + tx];
    else ds_A[ty][tx] = 0.0;

    if (Col < N && phase * TILE_WIDTH + ty < dim) ds_B[ty][tx] = X_t[(phase * TILE_WIDTH + ty) * N + Col];
    else ds_B[ty][tx] = 0.0;

    __syncthreads();

    if (Row < N && Col < N) {
        for (int ii = 0; ii < TILE_WIDTH; ii++) {
            double t = ds_A[ty][ii] - ds_B[ii][tx];
            Cvalue += t * t;
        }
    }

    __syncthreads();
  }

  if (Row < N && Col < N) D[Row * N + Col] = sqrt(Cvalue), idxMat[Row * N + Col] = Col;
}

__global__ void oddEvenSort(double *values, int *keys, int t, int size) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if(!t && idx * 2 + 1 < size) {
    if(values[idx * 2] > values[idx * 2 + 1]) {
      double tmp = values[idx * 2];
      values[idx * 2] = values[idx * 2 + 1];
      values[idx * 2 + 1] = tmp;
      int i = keys[idx * 2];
      keys[idx * 2] = keys[idx * 2 + 1];
      keys[idx * 2 + 1] = i;
    }
  }
  else if(idx * 2 + 2 < size){
    if(values[idx * 2 + 1] > values[idx * 2 + 2]) {
      double tmp = values[idx * 2 + 1];
      values[idx * 2 + 1] = values[idx * 2 + 2];
      values[idx * 2 + 2] = tmp;
      int i = keys[idx * 2 + 1];
      keys[idx * 2 + 1] = keys[idx * 2 + 2];
      keys[idx * 2 + 2] = i;
    }
  }
}

int main(int argc, char *argv[]) {
  assert(argc == 6);

  int N = atoi(argv[1]);
  int dim = atoi(argv[2]);
  int K = atoi(argv[3]);
  int *IdxMat;
  int *dIdxMat;
  double *X;
  double *X_t;
  double *dX;
  double *dX_t;
  double *D;
  double *dD;
  string title;
  clock_t p;

  CUDACheck(hipHostMalloc((void **)&X, N * dim * sizeof(double), hipHostMallocDefault));
  CUDACheck(hipHostMalloc((void **)&X_t, dim * N * sizeof(double), hipHostMallocDefault));
  CUDACheck(hipHostMalloc((void **)&D, N * N * sizeof(double), hipHostMallocDefault));
  CUDACheck(hipHostMalloc((void **)&IdxMat, N * N * sizeof(int), hipHostMallocDefault));

  CUDACheck(hipMalloc((void **)&dX, N * dim * sizeof(double)));
  CUDACheck(hipMalloc((void **)&dX_t, dim * N *sizeof(double)));
  CUDACheck(hipMalloc((void **)&dD, N * N * sizeof(double)));
  CUDACheck(hipMalloc((void **)&dIdxMat, N * N * sizeof(int)));
  
  cout << name << endl;
  cout << "N=" << N << " dim=" << dim << " K=" << K << " " << argv[4] << endl;

  p = clock();

  ifstream fin;
  fin.open(argv[4]);

  fin >> title;

  for(int i = 0; i < N; i++) {
    for(int j = 0; j < dim; j++) {
      fin >> X[i * dim + j];
      X_t[j * N + i] = X[i * dim + j];
    }
  }

  fin.close();
  
  cout << "step 0:Data Import::" << interval(&p) << "ms" << endl;

  CUDACheck(hipMemcpy(dX, X, N * dim * sizeof(double), hipMemcpyHostToDevice));
  CUDACheck(hipMemcpy(dX_t, X_t, dim * N * sizeof(double), hipMemcpyHostToDevice));

  cout << "step 1:Memcpy H2D::" << interval(&p) << "ms" << endl;

  dim3 grid_1((N - 1) / TILE_WIDTH + 1, (N - 1) / TILE_WIDTH + 1, 1);
  dim3 block_1(TILE_WIDTH, TILE_WIDTH, 1);
  size_t shm_size = 2 * TILE_WIDTH * TILE_WIDTH * sizeof(double);
  matrixDistance<<<grid_1, block_1, shm_size>>>(dX, dX_t, dD, dIdxMat, N, dim);

  CUDACheck(hipDeviceSynchronize());

  cout << "step 2:Distance::" << interval(&p) << "ms" << endl;

  dim3 grid_2((N / 2 - 1) / THREADS_PER_BLOCK + 1, 1, 1);
  dim3 block_2(THREADS_PER_BLOCK, 1, 1);

  for(int i = 0; i < N; i++) {
    for(int j = 0; j < N; j++) oddEvenSort<<<grid_2, block_2>>>(dD + i * N, dIdxMat + i * N, j % 2, N);
  }

  CUDACheck(hipDeviceSynchronize());

  cout << "step 3:Sort::" << interval(&p) << "ms" << endl;

  CUDACheck(hipMemcpy(IdxMat, dIdxMat, N * N * sizeof(int), hipMemcpyDeviceToHost));

  cout << "step 4:Memcpy D2H::" << interval(&p) << "ms" << endl;

  ofstream fout;
  fout.open(string(argv[5]));

  for(int i = 0; i < N; i++) {
    // omit first one -> i-i pair
    for(int j = 1; j < K + 1; j++) fout << IdxMat[i * N + j] << " ";

    fout << endl;
  }

  fout.close();

  cout << "step 5:Export Result::" << interval(&p) << "ms" << endl;

  hipHostFree(X);
  hipHostFree(X_t);
  hipHostFree(D);
  hipHostFree(IdxMat);

  hipFree(dX);
  hipFree(dX_t);
  hipFree(dD);
  hipFree(dIdxMat);
}